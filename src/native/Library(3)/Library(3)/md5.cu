#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "md5.h"

// Constants for MD5
__device__ const unsigned int k[] = {
    0xd76aa478, 0xe8c7b756, 0x242070db, 0xc1bdceee,
    0xf57c0faf, 0x4787c62a, 0xa8304613, 0xfd469501,
    0x698098d8, 0x8b44f7af, 0xffff5bb1, 0x895cd7be,
    0x6b901122, 0xfd987193, 0xa679438e, 0x49b40821,
    0xf61e2562, 0xc040b340, 0x265e5a51, 0xe9b6c7aa,
    0xd62f105d, 0x02441453, 0xd8a1e681, 0xe7d3fbc8,
    0x21e1cde6, 0xc33707d6, 0xf4d50d87, 0x455a14ed,
    0xa9e3e905, 0xfcefa3f8, 0x676f02d9, 0x8d2a4c8a,
    0xfffa3942, 0x8771f681, 0x6d9d6122, 0xfde5380c,
    0xa4beea44, 0x4bdecfa9, 0xf6bb4b60, 0xbebfbc70,
    0x289b7ec6, 0xeaa127fa, 0xd4ef3085, 0x04881d05,
    0xd9d4d039, 0xe6db99e5, 0x1fa27cf8, 0xc4ac5665,
    0xf4292244, 0x432aff97, 0xab9423a7, 0xfc93a039,
    0x655b59c3, 0x8f0ccc92, 0xffeff47d, 0x85845dd1,
    0x6fa87e4f, 0xfe2ce6e0, 0xa3014314, 0x4e0811a1,
    0xf7537e82, 0xbd3af235, 0x2ad7d2bb, 0xeb86d391
};

// Per-round shift amounts
__device__ const unsigned int r[] = {
     7, 12, 17, 22,  7, 12, 17, 22,
     7, 12, 17, 22,  7, 12, 17, 22,
     5,  9, 14, 20,  5,  9, 14, 20,
     5,  9, 14, 20,  5,  9, 14, 20,
     4, 11, 16, 23,  4, 11, 16, 23,
     4, 11, 16, 23,  4, 11, 16, 23,
     6, 10, 15, 21,  6, 10, 15, 21,
     6, 10, 15, 21,  6, 10, 15, 21
};

__device__ unsigned int leftrotate(unsigned int x, unsigned int c) {
    return (x << c) | (x >> (32 - c));
}

__device__ void md5(const char* initial_msg, char* output) {
    int len = 0;
    while (initial_msg[len] != '\0') len++;

    unsigned int a0 = 0x67452301;
    unsigned int b0 = 0xefcdab89;
    unsigned int c0 = 0x98badcfe;
    unsigned int d0 = 0x10325476;

    unsigned int msg[16] = { 0 };
    for (int i = 0; i < len; ++i) {
        msg[i >> 2] |= ((unsigned int)(unsigned char)initial_msg[i]) << ((i % 4) * 8);
    }
    msg[len >> 2] |= 0x80 << ((len % 4) * 8);
    msg[14] = len * 8;

    unsigned int A = a0;
    unsigned int B = b0;
    unsigned int C = c0;
    unsigned int D = d0;

    for (int i = 0; i < 64; ++i) {
        unsigned int F, g;

        if (i < 16) {
            F = (B & C) | ((~B) & D);
            g = i;
        }
        else if (i < 32) {
            F = (D & B) | ((~D) & C);
            g = (5 * i + 1) % 16;
        }
        else if (i < 48) {
            F = B ^ C ^ D;
            g = (3 * i + 5) % 16;
        }
        else {
            F = C ^ (B | (~D));
            g = (7 * i) % 16;
        }

        unsigned int temp = D;
        D = C;
        C = B;
        B = B + leftrotate((A + F + k[i] + msg[g]), r[i]);
        A = temp;
    }

    a0 += A;
    b0 += B;
    c0 += C;
    d0 += D;

    sprintf(output, "%08x%08x%08x%08x", a0, b0, c0, d0);
}
